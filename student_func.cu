#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"
#include <stdio.h>
#include <thrust/extrema.h>
// #include "reference_calc.h"
#define maxThreadsPerBlock  1024
#define itemsPerThread  16
#define binsPerThread  8
#define blockSize_x  64
#define blockSize_y  maxThreadsPerBlock/blockSize_x
#define bitMaxThreadsPerBlock 10



__global__ void shmem_reduce_min_kernel(float * d_out, const float * d_in,const int size)
{
    // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;
    unsigned int s = blockDim.x / 2;
    // load shared mem from global mem
    if (tid < s) 
    {
      sdata[tid] = min(d_in[(myId < size) ? myId : (size - 1)],d_in[((myId + s) < size) ? (myId + s) : (size - 1)]);
    }
    else
    {
      return;
    }
    __syncthreads();            // make sure entire block is loaded!

    // do reduction in shared mem
    for (s >>= 1; s > 1; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] = min(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = min(sdata[0], sdata[1]);
    }
}

__global__ void shmem_reduce_max_kernel(float * d_out, const float * d_in,const int size)
{
    // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;
    unsigned int s = blockDim.x / 2;
    // load shared mem from global mem
    if (tid < s) 
    {
      sdata[tid] = max(d_in[(myId < size) ? myId : (size - 1)],
                       d_in[((myId + s) < size) ? (myId + s) : (size - 1)]);
    }
    else
    {
      return;
    }
    __syncthreads();            // make sure entire block is loaded!

    // do reduction in shared mem
    for (s >>= 1; s > 1; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] = max(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();        // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = max(sdata[0], sdata[1]);
    }
}

void min_max_finding(float * min_logLum, 
                     float * max_logLum, 
                     const float* const d_in, 
                     int size)
{
    // assumes that size is not greater than maxThreadsPerBlock^2
    // and that size is a multiple of maxThreadsPerBlock

float * d_out_min, * d_out_max;


checkCudaErrors(hipMalloc((void **) &d_out_min, sizeof(float)));
checkCudaErrors(hipMalloc((void **) &d_out_max, sizeof(float)));


    int blocks = (size / maxThreadsPerBlock) + ((size % maxThreadsPerBlock == 0) ? 0 : 1);
    int size_ = size;
    float *d_intermediate;
    float *d_intermediate2;
    if(blocks == 1)
    {
      shmem_reduce_min_kernel<<<1, maxThreadsPerBlock, maxThreadsPerBlock * sizeof(float) / 2>>>
        (d_out_min, d_in, size_);
      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    }
    else
    {
      checkCudaErrors(hipMalloc(&d_intermediate,(size_t) blocks * sizeof(float)));
      shmem_reduce_min_kernel<<<blocks, maxThreadsPerBlock, maxThreadsPerBlock * sizeof(float) / 2>>>
        (d_intermediate, d_in, size_);
      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
      size_ = blocks;
      blocks = (blocks >> bitMaxThreadsPerBlock) + 1;
      if(blocks == 1)
      {
        shmem_reduce_min_kernel<<<1, maxThreadsPerBlock, maxThreadsPerBlock * sizeof(float) / 2>>>
          (d_out_min, d_intermediate, size_);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
      }
      else 
      {
        checkCudaErrors(hipMalloc(&d_intermediate2,(size_t) blocks * sizeof(float)));
        shmem_reduce_min_kernel<<<blocks, maxThreadsPerBlock, maxThreadsPerBlock * sizeof(float) / 2>>>
          (d_intermediate2, d_intermediate, size_); 
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
        shmem_reduce_min_kernel<<<1, maxThreadsPerBlock, maxThreadsPerBlock * sizeof(float) / 2>>>
          (d_out_min, d_intermediate2, blocks); 
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
      } 
    }
  
    blocks = (size >> bitMaxThreadsPerBlock) + 1;
    if(blocks == 1)
    {
      shmem_reduce_max_kernel<<<1, maxThreadsPerBlock, maxThreadsPerBlock * sizeof(float) / 2>>>
        (d_out_max, d_in, size);
      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
    }
    else
    {
      shmem_reduce_max_kernel<<<blocks, maxThreadsPerBlock, maxThreadsPerBlock * sizeof(float) / 2>>>
        (d_intermediate, d_in, size);
      hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
      size = blocks;
      blocks = (blocks >> bitMaxThreadsPerBlock) + 1;
      if(blocks == 1)
      {
        shmem_reduce_max_kernel<<<1, maxThreadsPerBlock, maxThreadsPerBlock * sizeof(float) / 2>>>
          (d_out_max, d_intermediate, size);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipFree(d_intermediate));
      }
      else 
      {
        shmem_reduce_max_kernel<<<blocks, maxThreadsPerBlock, maxThreadsPerBlock * sizeof(float) / 2>>>
          (d_intermediate2, d_intermediate, size); 
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
        shmem_reduce_max_kernel<<<1, maxThreadsPerBlock, maxThreadsPerBlock * sizeof(float) / 2>>>
          (d_out_max, d_intermediate2, blocks);
        hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipFree(d_intermediate));
        checkCudaErrors(hipFree(d_intermediate2));
      } 
    } 

  checkCudaErrors(hipMemcpy(min_logLum, d_out_min, sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(max_logLum, d_out_max, sizeof(float), hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(d_out_min));
  checkCudaErrors(hipFree(d_out_max));
}

__global__ void total_atomic_histo(int *d_bins, const float *d_in, const float min_logLum,
                                   const float max_logLum, const int numBins, const int size)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    if(myId >= size) return;
    unsigned int bin = min(static_cast<unsigned int>(numBins - 1),
                           static_cast<unsigned int>((d_in[myId] - min_logLum) / (max_logLum - min_logLum) * numBins));
    atomicAdd(&(d_bins[bin]), 1);
}

__global__ void reduce_atomic_histo(int *d_bins, const float *d_in, const float min_logLum,
                                   const float max_logLum, const int numBins, const int size)
{
  extern __shared__ int sdataint[];
  int myIdx_x = threadIdx.x + blockDim.x * blockIdx.x;
  int myIdx_y = threadIdx.y + blockDim.y * blockIdx.y;
  int i, tid = threadIdx.x + blockDim.x * threadIdx.y;
  int bin, s;
  int itemIdx0 = myIdx_x * itemsPerThread,
      binIdx0  = myIdx_y * binsPerThread;
  int localHisto[binsPerThread];

  for(i = 0; i < binsPerThread; i++) localHisto[i] = 0;

  if(itemIdx0 < size)
  {
    for(i = 0; itemIdx0 + i < size && i < itemsPerThread; i++)
    {
      bin = static_cast<unsigned int>((d_in[itemIdx0 + i] - min_logLum) 
        / (max_logLum - min_logLum) * numBins);
      bin = (bin == numBins) ? bin - 1 : bin;
      if(bin >= binIdx0 && bin < (binIdx0 + binsPerThread))
      localHisto[bin % binsPerThread] = localHisto[bin % binsPerThread] + 1;
    }
  }


  for(i = 0; i < binsPerThread; i++)
  {
    s = blockDim.x;
    sdataint[tid] = localHisto[i];
    __syncthreads();           
    for (s >>= 1; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            sdataint[tid] = sdataint[tid] + sdataint[tid + s];
        }
        __syncthreads();        
    }
    if(threadIdx.x == 0 && binIdx0 + i < numBins) atomicAdd(&(d_bins[binIdx0 + i]), sdataint[blockDim.x * threadIdx.y]);
    __syncthreads();
  }
}


void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{

int size = numRows * numCols;

min_max_finding(&min_logLum, &max_logLum, d_logLuminance, size);

int *d_bins;

checkCudaErrors(hipMalloc((void **) &d_bins, sizeof(int) * numBins));
checkCudaErrors(hipMemset(d_bins, 0, sizeof(int) * numBins));

// totally done by atomic add

int blocks = (size / maxThreadsPerBlock) + ((size % maxThreadsPerBlock == 0) ? 0 : 1);

total_atomic_histo<<<blocks,maxThreadsPerBlock>>>(d_bins, d_logLuminance, min_logLum,
                   max_logLum, numBins, size);

// first reduce then atomic add

int numBlocks_x = (size / (blockSize_x * itemsPerThread)) + ((size % (blockSize_x * itemsPerThread) == 0) ? 0 : 1);
int numBlocks_y = (numBins / (blockSize_y * binsPerThread)) + ((numBins % (blockSize_y * binsPerThread) == 0) ? 0 : 1);
dim3 grid(numBlocks_x, numBlocks_y);
dim3 block(blockSize_x, blockSize_y);

reduce_atomic_histo<<<grid, block, maxThreadsPerBlock * sizeof(int)>>>(d_bins, d_logLuminance, min_logLum,
                   max_logLum, numBins, size);


int histo[numBins];
int h_cdf[numBins];
checkCudaErrors(hipMemcpy(histo,d_bins, sizeof(int) * numBins,hipMemcpyDeviceToHost));

  h_cdf[0] = 0;
  for (size_t i = 1; i < numBins; ++i) {
    h_cdf[i] = h_cdf[i - 1] + histo[i - 1];
  }
checkCudaErrors(hipMemcpy(d_cdf,h_cdf, sizeof(int) * numBins,hipMemcpyHostToDevice));

checkCudaErrors(hipFree(d_bins));


  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */


}
